
#include <hip/hip_runtime.h>
#ifdef DFTEFE_WITH_DEVICE_CUDA
#  include <utils/DeviceKernelLauncher.h>
#  include <utils/DeviceDataTypeOverloads.cuh>
#  include <utils/DataTypeOverloads.h>
#  include <utils/MemoryTransfer.h>
#  include <utils/Exceptions.h>
#  include <linearAlgebra/BlasLapackKernels.h>
#  include <complex>
#  include <algorithm>
namespace dftefe
{
  namespace linearAlgebra
  {
    namespace blasLapack
    {
      namespace
      {
        template <typename ValueType1, typename ValueType2, typename ValueType3>
        __global__ void
        ascaleDeviceKernel(const size_type   size,
                           const ValueType1  alpha,
                           const ValueType2 *x,
                           ValueType3 *      z)
        {
          const size_type globalThreadId =
            blockIdx.x * blockDim.x + threadIdx.x;
          for (size_type i = globalThreadId; i < size;
               i += blockDim.x * gridDim.x)
            {
              z[i] = dftefe::utils::mult(alpha, x[i]);
            }
        }

        template <typename ValueType1, typename ValueType2, typename ValueType3>
        __global__ void
        hadamardProductDeviceKernel(const size_type   size,
                                    const ValueType1 *x,
                                    const ValueType2 *y,
                                    ValueType3 *      z)
        {
          const size_type globalThreadId =
            blockIdx.x * blockDim.x + threadIdx.x;
          for (size_type i = globalThreadId; i < size;
               i += blockDim.x * gridDim.x)
            {
              z[i] = dftefe::utils::mult(x[i], y[i]);
            }
        }

        template <typename ValueType1, typename ValueType2, typename ValueType3>
        __global__ void
        axpbyDeviceKernel(const size_type   size,
                          const ValueType3  alpha,
                          const ValueType1 *x,
                          const ValueType3  beta,
                          const ValueType2 *y,
                          ValueType3 *      z)
        {
          const size_type globalThreadId =
            blockIdx.x * blockDim.x + threadIdx.x;
          for (size_type i = globalThreadId; i < size;
               i += blockDim.x * gridDim.x)
            {
              z[i] = dftefe::utils::add(dftefe::utils::mult(alpha, x[i]),
                                        dftefe::utils::mult(beta, y[i]));
            }
        }


        template <typename ValueType>
        __global__ void
        absSquareEntriesDeviceKernel(const size_type  size,
                                     const ValueType *x,
                                     double *         y)
        {
          const size_type globalThreadId =
            blockIdx.x * blockDim.x + threadIdx.x;
          for (size_type i = globalThreadId; i < size;
               i += blockDim.x * gridDim.x)
            {
              const double temp = dftefe::utils::abs(x[i]);
              y[i]              = temp * temp;
            }
        }

      } // namespace


      template <typename ValueType1, typename ValueType2>
      void
      Kernels<dftefe::utils::MemorySpace::DEVICE, ValueType1, ValueType2>::
        ascale(const size_type                      size,
               const ValueType1                     alpha,
               const ValueType2 *                   x,
               scalar_type<ValueType1, ValueType2> *z)
      {
        ascaleDeviceKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                             dftefe::utils::BLOCK_SIZE>>>(
          size,
          dftefe::utils::makeDataTypeDeviceCompatible(alpha),
          dftefe::utils::makeDataTypeDeviceCompatible(x),
          dftefe::utils::makeDataTypeDeviceCompatible(z));
      }

      template <typename ValueType1, typename ValueType2>
      void
      Kernels<dftefe::utils::MemorySpace::DEVICE, ValueType1, ValueType2>::
        hadamardProduct(const size_type                      size,
                        const ValueType1 *                   x,
                        const ValueType2 *                   y,
                        scalar_type<ValueType1, ValueType2> *z)
      {
        hadamardProductDeviceKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                                      dftefe::utils::BLOCK_SIZE>>>(
          size,
          dftefe::utils::makeDataTypeDeviceCompatible(x),
          dftefe::utils::makeDataTypeDeviceCompatible(y),
          dftefe::utils::makeDataTypeDeviceCompatible(z));
      }

      template <typename ValueType1, typename ValueType2>
      void
      Kernels<dftefe::utils::MemorySpace::DEVICE, ValueType1, ValueType2>::
        axpby(const size_type                           size,
              const scalar_type<ValueType1, ValueType2> alpha,
              const ValueType1 *                        x,
              const scalar_type<ValueType1, ValueType2> beta,
              const ValueType2 *                        y,
              scalar_type<ValueType1, ValueType2> *     z)
      {
        axpbyDeviceKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                            dftefe::utils::BLOCK_SIZE>>>(
          size,
          dftefe::utils::makeDataTypeDeviceCompatible(alpha),
          dftefe::utils::makeDataTypeDeviceCompatible(x),
          dftefe::utils::makeDataTypeDeviceCompatible(beta),
          dftefe::utils::makeDataTypeDeviceCompatible(y),
          dftefe::utils::makeDataTypeDeviceCompatible(z));
      }


      template <typename ValueType1, typename ValueType2>
      std::vector<double>
      Kernels<dftefe::utils::MemorySpace::DEVICE, ValueType1, ValueType2>::
        amaxsMultiVector(size_type         vecSize,
                         size_type         numVec,
                         ValueType1 const *multiVecData)
      {
        std::vector<double> amaxs(numVec, 0);

        utils::throwException(
          false,
          "amaxsMultiVector() is not implemented for dftefe::utils::MemorySpace::DEVICE .... ");
        return amaxs;
      }



      template <typename ValueType1, typename ValueType2>
      std::vector<double>
      Kernels<dftefe::utils::MemorySpace::DEVICE, ValueType1, ValueType2>::
        nrms2MultiVector(
          size_type                                      vecSize,
          size_type                                      numVec,
          ValueType1 const *                             multiVecData,
          BlasQueue<dftefe::utils::MemorySpace::DEVICE> &BlasQueue)
      {
        std::vector<double> nrms2(numVec, 0);

        dftefe::utils::MemoryStorage<double, dftefe::utils::MemorySpace::DEVICE>
          nrmsSqVecDevice(numVec, 0.0);
        dftefe::utils::MemoryStorage<double, dftefe::utils::MemorySpace::DEVICE>
          onesVecDevice(vecSize, 1.0);
        dftefe::utils::MemoryStorage<double, dftefe::utils::MemorySpace::DEVICE>
          squaredEntriesDevice(vecSize * numVec, 0.0);

        absSquareEntriesDeviceKernel<<<
          (vecSize * numVec) / dftefe::utils::BLOCK_SIZE + 1,
          dftefe::utils::BLOCK_SIZE>>>(
          vecSize * numVec,
          dftefe::utils::makeDataTypeDeviceCompatible(multiVecData),
          dftefe::utils::makeDataTypeDeviceCompatible(
            squaredEntriesDevice.begin()));

        blas::gemm(Layout::ColMajor,
                   Op::NoTrans,
                   Op::Trans,
                   1,
                   numVec,
                   vecSize,
                   1.0,
                   onesVecDevice.data(),
                   1,
                   squaredEntriesDevice.data(),
                   numVec,
                   1.0,
                   nrmsSqVecDevice.data(),
                   1,
                   BlasQueue);


        nrmsSqVecDevice.copyTo<dftefe::utils::MemorySpace::DEVICE>(&nrms2[0]);

        for (size_type i = 0; i < numVec; i++)
          nrms2[i] = std::sqrt(nrms2[i]);

        return nrms2;
      }

#  define EXPLICITLY_INSTANTIATE(T1, T2, M) template class Kernels<M, T1, T2>;



      EXPLICITLY_INSTANTIATE(float, float, dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(float, double, dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(float,
                             std::complex<float>,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(float,
                             std::complex<double>,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(double, float, dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(double,
                             double,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(double,
                             std::complex<float>,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(double,
                             std::complex<double>,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(std::complex<float>,
                             float,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(std::complex<float>,
                             double,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(std::complex<float>,
                             std::complex<float>,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(std::complex<float>,
                             std::complex<double>,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(std::complex<double>,
                             float,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(std::complex<double>,
                             double,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(std::complex<double>,
                             std::complex<float>,
                             dftefe::utils::MemorySpace::DEVICE);
      EXPLICITLY_INSTANTIATE(std::complex<double>,
                             std::complex<double>,
                             dftefe::utils::MemorySpace::DEVICE);
    } // namespace blasLapack
  }   // namespace linearAlgebra
} // namespace dftefe
#endif
